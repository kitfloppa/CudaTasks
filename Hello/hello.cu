#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void hello() {
    printf("Hello CUDA from GPU!!!\n");
}

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    hello<<<1, 1>>>();
    hipDeviceSynchronize();

    printf("Hello CPU\n");
    printf("Device name: %s", prop.name);
    
    return 0;
} // nvcc -arch=sm_35 hello.cu -o hello
