#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>

__global__ void monte_carlo(float *res, double *mass_1, double *mass_2, int n) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;

    double x = mass_1[tid];
    double y = mass_2[tid];

    if (((x * x) + (y * y)) <= 1) atomicAdd(res, 1.0);
}

int main() {
    int n = 10000000, block = 256, numblock = (n + block - 1) / block;
    float *result, res = 0;
    double *mass_1, *mass_2, pi = 0;

    hipMalloc((void**)&result, sizeof(float));
    hipMalloc((void**)&mass_1, n * sizeof(double));
    hipMalloc((void**)&mass_2, n * sizeof(double));

    hiprandGenerator_t ran;
    hiprandCreateGenerator(&ran, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(ran, 1234ULL^time(NULL));
    
    hiprandGenerateUniformDouble(ran, mass_1, n);
    hiprandGenerateUniformDouble(ran, mass_2, n);

    monte_carlo<<<numblock, block>>>(result, mass_1, mass_2, n);
    hipMemcpy(&res, result, sizeof(int), hipMemcpyDeviceToHost);
    pi = 4 * res / n;

    hipFree(result);
    hipFree(mass_1);
    hipFree(mass_2);
    printf("PI = %f", pi);

    return 0;
} // nvcc monte_carlo_atomic.cu -o monte_carlo_atomic -l hiprand
