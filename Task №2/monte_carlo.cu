#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#include <hiprand/hiprand.h>
#include <iostream>
#include <numeric>

__global__ void monte_carlo(double *x, double *y, double *res, unsigned int n) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;

    if ((x[tid] * x[tid]) + (y[tid] * y[tid]) <= 1) res[tid] = 1;
	else res[tid] = 0;
}

int main() {
    const unsigned int n = 5000000, block = 256, numblocks = (n + block - 1) / block;
    double *result = (double*)malloc(n * sizeof(double)); 
    double *x, *y, *mass_c, res = 0, pi = 0;

	hipMalloc(&x, n * sizeof(double));
	hipMalloc(&y, n * sizeof(double));
	hipMalloc(&mass_c, n * sizeof(double));

    hiprandGenerator_t ran;
	hiprandCreateGenerator(&ran, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(ran, 1234ULL);

    hiprandGenerateUniformDouble(ran, x, n);
	hiprandGenerateUniformDouble(ran, y, n);

    monte_carlo<<<numblocks, block>>>(x, y, mass_c, n);
    hipMemcpy(result, mass_c, n * sizeof(double), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < n; ++i) res += result[i];
    pi = 4 * res / n;

    hipFree(x);
    hipFree(y);
    hipFree(mass_c);
    printf("Pi = %f", pi);

    return 0;
}