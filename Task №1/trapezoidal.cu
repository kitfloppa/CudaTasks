#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void trapezoidalIntegral(double *sum, double h, double begin) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    double x_1 = begin + i * h, x_2 = begin + (i + 1) * h;
	double x_3 = x_1 * x_1, x_4 = x_2 * x_2;

    sum[i] = 0.5 * (x_2 - x_1) * (x_3 + x_4);
}

__global__ void reduce(double *res) {
	unsigned int tid = threadIdx.x;

	for (size_t k = 1; k < blockDim.x; k *= 2) {
		unsigned int index = 2 * k * tid;

		if (index < blockDim.x) res[index] += res[index + k];
		__syncthreads();
	}
}

int main() {
    const double a = 0, b = 15;
    const int n = 1000;
    double h = (b - a) / n, result = 0;
    double *mas_c;

    hipMalloc((void**)&mas_c, n * sizeof(double));
	
	trapezoidalIntegral<<<1, n>>>(mas_c, h, a);
	reduce<<<1, n>>>(mas_c);

	hipMemcpy(&result, mas_c, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(mas_c);
	printf("Result = %f", result);
    
    return 0;
} // nvcc trapezoidal.cu -o trapezoidal
