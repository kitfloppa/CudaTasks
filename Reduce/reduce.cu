#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define MAX_ERR 1e-6

__global__ void vector_add(double *res, double *a, double *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) res[tid] = a[tid] + b[tid];
}

__global__ void reduce(double *res) {
	unsigned int tid = threadIdx.x;

	for (size_t k = blockDim.x / 2; k > 0; k >>= 1) {
		if (tid < k) res[tid] += res[tid + k];
		__syncthreads();
	}
}

int main() {
    int n = 256, grid_size = ((n + n) / n);;
    double *a, *b, *c, sum = 0;
    double *d_a, *d_b, *d_c; 

    a = (double*)malloc(n * sizeof(double));
    b = (double*)malloc(n * sizeof(double));
    c = (double*)malloc(n * sizeof(double));

    for(size_t i = 0; i < n; ++i){
        a[i] = 1010.0;
        b[i] = 1011.0;
    }

    hipMalloc((void**)&d_a, n * sizeof(double));
    hipMalloc((void**)&d_b, n * sizeof(double));
    hipMalloc((void**)&d_c, n * sizeof(double));

    hipMemcpy(d_a, a, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);

    vector_add<<<grid_size, n>>>(d_c, d_a, d_b, n);
    hipMemcpy(c, d_c, n * sizeof(double), hipMemcpyDeviceToHost);

    for(size_t i = 0; i < n; ++i) assert(fabs(c[i] - a[i] - b[i]) < MAX_ERR);

    reduce<<<grid_size, n>>>(d_c);
    hipMemcpy(&sum, d_c, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a); 
    free(b); 
    free(c);

    printf("Sum = %f", sum);

    return 0;
} // nvcc reduce.cu -o reduce
